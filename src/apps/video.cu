#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "config.h"
#include "video_fitter.h"
#include <experimental/filesystem>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/videoio.hpp>

#include <string>
#include <cmath>
#include <deque>

#include <vector>
#include <stdio.h>
#include <numeric>
#include <random>
#include <algorithm>    // std::shuffle
#include <set>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "constants.h"
#include "renderer.h"

#include "derivative_computer.h"
#include "logbarrier_initializer.h"

#include "model_fitter.h"
#include "funcs.h"
#include "newfuncs.h"
#include "rotation_header.h"
#include "Optimizer.h"
#include "camera.h"
#include "solver.h"
#include "preprocessing.h"



#include <glob.h> // glob(), globfree()
#include <string.h> // memset()
#include <stdexcept>
#include <sstream>

#include <opencv2/dnn.hpp>

#ifdef VISUALIZE_3D
#include "GLfuncs.h"

#endif


using namespace cv;
using namespace cv::dnn;

using std::vector;

// these exist on the GPU side
texture<float,2> EX_texture;
texture<float,2> EY_texture;
texture<float,2> EZ_texture;

texture<float,2> IX_texture;
texture<float,2> IY_texture;
texture<float,2> IZ_texture;

texture<float,2> TEX_texture;

using std::vector;

int create_data_for_multiframe(const std::string& imdir, Renderer &r, const std::string& outdir, const uint subj_id, float fovx, float fovy,
                               vector<vector<float> >& xps, vector<vector<float> >& yps,
                               vector<vector<float> >& xranges, vector<vector<float> >& yranges,
                               vector<Mat> &selected_frames, vector<std::string>& result_basepaths, const std::vector<int> &angle_idx,
                               Net &detection_net, Net &landmark_net, Net &leye_net, Net &reye_net, Net &mouth_net, Net &correction_net,
                               bool set_RESIZE_COEF_via_median=true, int combination_id = -1);

//const int KERNEL_RADIUS=2;

// /media/v/SSD1TB/dataset/videos/treecam/ML/ML0001.mp4 /media/v/SSD1TB/dataset/videos/treecam/ML/ML0001.mp4.avi
// /media/v/SSD1TB/dataset/Florence/images/for_experiments/ /media/v/SSD1TB/dataset/Florence/results/3DIv2

// /media/v/SSD1TB/dataset/BU4DFE/images/ /media/v/SSD1TB/dataset/BU4DFE/results/3DIv2
///data/videos/treecam/1041a/test_RA2_NA.mkv /data/videos/treecam/ML/output/ 15


int main(int argc, char** argv)
{
    ///////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////

    //    std::string filepath = "/data/videos/treecam/baby/bv1.mp4";
    // /media/v/Samsung_T5/dataset/videos/treecam/ML/ML0001.mp4 /media/v/Samsung_T5/dataset/videos/treecam/ML/ML0001.mp4.avi
    // /data/videos/treecam/ML/ML0001.mp4 /data/videos/treecam/ML/ML0001_TT.avi
    if (argc < 2) {
        std::cout << "You need at least one argument -- the filepath for the input video" << std::endl;
        return -1;
    }

    std::string config_filepath(argv[3]);
    config::set_params_from_YAML_file(config_filepath);

    std::string filepath(argv[1]);

    std::string outputVideoPath("output.avi");
    if (argc < 3) {
        std::cout << "we need at least 2 arguments (the 2nd needs to be output dir)" << std::endl;
    }

    std::string outputVideoDir = argv[2];

    if (!std::experimental::filesystem::exists(outputVideoDir))
        std::experimental::filesystem::create_directory(outputVideoDir);

    if (config::OUTDIR_WITH_PARAMS)
        outputVideoDir += "/" + config::get_key();
    else
        outputVideoDir += "/";

    if (!std::experimental::filesystem::exists(outputVideoDir))
        std::experimental::filesystem::create_directory(outputVideoDir);

    Camera cam0;
    float field_of_view = 40;

    //    std::string calibration_path = "./models/cameras/TreeCam_1041a.txt";
    std::string calibration_path("");
    if (argc == 5) {
        if (!is_float(argv[4]))
        {
            calibration_path = argv[4];
            cam0.init(calibration_path);
        } else {
            field_of_view = std::stof(argv[4]);

            outputVideoDir += std::string("/") + argv[4];

            if (!std::experimental::filesystem::exists(outputVideoDir))
                std::experimental::filesystem::create_directory(outputVideoDir);
        }
    }


    outputVideoPath = outputVideoDir + "/" + remove_extension(base_name(filepath)) + ".avi";


    std::string identityPath = remove_extension(outputVideoPath) + ".id.txt";
    std::string texturePath = remove_extension(outputVideoPath) + ".tex.txt";

    /*
    if (std::experimental::filesystem::exists(identityPath))
        return 0;
        */

    if (!cam0.initialized)
    {
        cv::VideoCapture tmpCap(filepath);

        int video_width = tmpCap.get(cv::CAP_PROP_FRAME_WIDTH);
        int video_height = tmpCap.get(cv::CAP_PROP_FRAME_HEIGHT);

        if (config::PRINT_DEBUG)
            std::cout << video_width << '\t' << video_height << std::endl;

        tmpCap.release();

        float cam_cx = video_width/2.0;
        float cam_cy = video_height/2.0;

//        double angle_x = 120.0f*M_PI/180.0; // angle in radians
        double angle_x = field_of_view*M_PI/180.0; // angle in radians
        double angle_y = angle_x; //60.0f*M_PI/180.0; //(cam_cy/cam_cx)*angle_x;

        float cam_alphax = cam_cx/(tan(angle_x/2.0));
        float cam_alphay = cam_alphax; //cam_cy/(tan(angle_y/2.0));

        /*
        std::cout << "cam_alphax: " << cam_alphax << std::endl;
        std::cout << "cam_alphay: " << cam_alphay << std::endl;
        */

        cam0.init(cam_alphax, cam_alphay, cam_cx, cam_cy, false);
    }





    float *h_X0, *h_Y0, *h_Z0, *h_tex_mu;
    h_X0 = (float*)malloc( NPTS*sizeof(float) );
    h_Y0 = (float*)malloc( NPTS*sizeof(float) );
    h_Z0 = (float*)malloc( NPTS*sizeof(float) );
    h_tex_mu = (float*)malloc( NPTS*sizeof(float) );

    {
        VideoFitter vf_identity(cam0,
                                NID_COEFS, NTEX_COEFS, config::K_EPSILON,
                                K_ALPHA_L, 0, config::K_EPSILON_L, config::NFRAMES,
                                false, false);



        hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
        hipChannelFormatDesc desc2 = hipCreateChannelDesc<float>();
        hipChannelFormatDesc desc3 = hipCreateChannelDesc<float>();

        // Start with expression bases
        hipBindTexture2D(0, EX_texture, vf_identity.r.d_EX_row_major, desc, config::K_EPSILON, NPTS, vf_identity.r.pitch);
        hipBindTexture2D(0, EY_texture, vf_identity.r.d_EY_row_major, desc, config::K_EPSILON, NPTS, vf_identity.r.pitch);
        hipBindTexture2D(0, EZ_texture, vf_identity.r.d_EZ_row_major, desc, config::K_EPSILON, NPTS, vf_identity.r.pitch);

        // Now identity bases
        if (vf_identity.r.use_identity)
        {
            hipBindTexture2D(0, IX_texture, vf_identity.r.d_IX_row_major, desc2, NID_COEFS, NPTS, vf_identity.r.pitch2);
            hipBindTexture2D(0, IY_texture, vf_identity.r.d_IY_row_major, desc2, NID_COEFS, NPTS, vf_identity.r.pitch2);
            hipBindTexture2D(0, IZ_texture, vf_identity.r.d_IZ_row_major, desc2, NID_COEFS, NPTS, vf_identity.r.pitch2);
        }

        // Finally the texture bases
        if (vf_identity.r.use_texture)
        {
            hipBindTexture2D(0, TEX_texture, vf_identity.r.d_TEX_row_major, desc3, NTEX_COEFS, NPTS, vf_identity.r.pitch3);
        }

        ///////////////////////////////////////////
        ///////////////////////////////////////////
        std::cout << "Learning the 3D identity of subject in video ... this may take a few minutes" << std::endl;
        vf_identity.learn_identity(filepath, h_X0, h_Y0, h_Z0, h_tex_mu);
        std::cout << "\tDone" << std::endl;
        ///////////////////////////////////////////
        ///////////////////////////////////////////


        if (vf_identity.r.use_identity) {
            hipUnbindTexture(IX_texture);
            hipUnbindTexture(IY_texture);
            hipUnbindTexture(IZ_texture);
        }

        if (vf_identity.r.use_texture) {
            hipUnbindTexture(TEX_texture);
        }

        hipUnbindTexture(EX_texture);
        hipUnbindTexture(EY_texture);
        hipUnbindTexture(EZ_texture);
    }




    if (config::OUTPUT_IDENTITY) {
        write_identity(identityPath, h_X0, h_Y0, h_Z0);
        write_texture(texturePath, h_tex_mu);
    }

    /*
    if (true) {

        free(h_X0);
        free(h_Y0);
        free(h_Z0);
        free(h_tex_mu);
        return 0;
    }
*/
    cam0.update_camera(1.0f);
    VideoFitter vf(cam0, 0, 0, config::K_EPSILON,
                   0, 0, config::K_EPSILON_L, config::TIME_T,
                    config::USE_TEMP_SMOOTHING, config::USE_EXP_REGULARIZATION,
                    h_X0, h_Y0, h_Z0, h_tex_mu);



    // Bind texture memories
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipChannelFormatDesc desc2 = hipCreateChannelDesc<float>();
    hipChannelFormatDesc desc3 = hipCreateChannelDesc<float>();

    // Start with expression bases
    hipBindTexture2D(0, EX_texture, vf.r.d_EX_row_major, desc, vf.r.Kepsilon, NPTS, vf.r.pitch);
    hipBindTexture2D(0, EY_texture, vf.r.d_EY_row_major, desc, vf.r.Kepsilon, NPTS, vf.r.pitch);
    hipBindTexture2D(0, EZ_texture, vf.r.d_EZ_row_major, desc, vf.r.Kepsilon, NPTS, vf.r.pitch);



    std::cout << "Processing video ..."  << std::endl;
    VideoOutput vid_out = vf.fit_video_frames_auto(filepath, outputVideoPath);
    std::cout << "\tDone" << std::endl;

    std::string outputVideoPath_3D(""), outputVideoPath_texture("");

    outputVideoPath_3D = outputVideoPath;
    outputVideoPath_3D.replace(outputVideoPath_3D.end()-4,outputVideoPath_3D.end(), "_3Drenders.avi");

    outputVideoPath_texture = outputVideoPath;
    outputVideoPath_texture.replace(outputVideoPath_texture.end()-4,outputVideoPath_texture.end(), "_texture.avi");

    std::string exp_path = outputVideoPath;
    exp_path.replace(exp_path.end()-4,exp_path.end(), ".expressions");

    std::string pose_path = outputVideoPath;
    pose_path.replace(pose_path.end()-4,pose_path.end(), ".poses");

    vid_out.save_expressions(exp_path);
    vid_out.save_poses(pose_path);

    if (config::OUTPUT_VISUALS)
    {
        std::cout << "Creating texture video ..." << std::endl;
        vf.visualize_texture(vid_out, filepath, outputVideoPath_texture);
        std::cout << "\tDone" << std::endl;

        std::cout << "Creating render video ..." << std::endl;
        vf.visualize_3dmesh(vid_out, filepath, outputVideoPath_3D);
        std::cout << "\tDone" << std::endl;
    }


    hipUnbindTexture(EX_texture);
    hipUnbindTexture(EY_texture);
    hipUnbindTexture(EZ_texture);


    free(h_X0);
    free(h_Y0);
    free(h_Z0);
    free(h_tex_mu);
}









__global__ void render_expression_basis_texture_colmajor_rotated2(
        const float* __restrict__ alphas, const float* __restrict__ betas, const float* __restrict__ gammas,
        const uint* __restrict__ indices, const int Nunique_pixels, const ushort* __restrict__ tl,
        float*  __restrict__ REX, float*  __restrict__ REY, float*  __restrict__ REZ,
        const ushort* __restrict__ triangle_idx, const float* R__)
{
    const int rowix = blockIdx.x;
    const int colix = threadIdx.x;
    __shared__ float R[9];

    if (colix < 9) {
        R[colix] = R__[colix];
    }
    __syncthreads();

    const float R00 = R[0]; const float R10 = R[1]; const float R20 = R[2];
    const float R01 = R[3]; const float R11 = R[4]; const float R21 = R[5];
    const float R02 = R[6]; const float R12 = R[7]; const float R22 = R[8];

    const int rel_index = indices[rowix];

    const int idx = threadIdx.x*Nredundant + blockIdx.x;

    const int tl_i1 = triangle_idx[rel_index];
    const int tl_i2 = tl_i1 + N_TRIANGLES;
    const int tl_i3 = tl_i2 + N_TRIANGLES;

    const float tmpx = tex2D(EX_texture,colix,tl[tl_i1])*alphas[rel_index] + tex2D(EX_texture,colix,tl[tl_i2])*betas[rel_index] + tex2D(EX_texture,colix,tl[tl_i3])*gammas[rel_index];
    const float tmpy = tex2D(EY_texture,colix,tl[tl_i1])*alphas[rel_index] + tex2D(EY_texture,colix,tl[tl_i2])*betas[rel_index] + tex2D(EY_texture,colix,tl[tl_i3])*gammas[rel_index];
    const float tmpz = tex2D(EZ_texture,colix,tl[tl_i1])*alphas[rel_index] + tex2D(EZ_texture,colix,tl[tl_i2])*betas[rel_index] + tex2D(EZ_texture,colix,tl[tl_i3])*gammas[rel_index];

    REX[idx] = tmpx*R00 + tmpy*R01 + tmpz*R02;
    REY[idx] = tmpx*R10 + tmpy*R11 + tmpz*R12;
    REZ[idx] = tmpx*R20 + tmpy*R21 + tmpz*R22;
}



__global__ void render_identity_basis_texture(
        const float* __restrict__ alphas, const float* __restrict__ betas, const float* __restrict__ gammas,
        const uint* __restrict__ indices, const int N1, const ushort* __restrict__ tl,
        float* __restrict__ RIX, float* __restrict__ RIY, float* __restrict__ RIZ,
        const ushort* __restrict__ triangle_idx, const ushort Kalpha)
{
    const int rowix = blockIdx.x;
    const int colix = threadIdx.x;

    const int rel_index = indices[rowix];

    //! Important! We fill REX, ... in a ROW-MAJOR order. This way it will be easier to extract a submatrix of REX that ignores the bottom of REX
    const int idx = threadIdx.x + Kalpha*blockIdx.x;

    const int tl_i1 = triangle_idx[rel_index];
    const int tl_i2 = tl_i1 + N_TRIANGLES;
    const int tl_i3 = tl_i2 + N_TRIANGLES;

    RIX[idx] = tex2D(IX_texture,colix,tl[tl_i1])*alphas[rel_index] + tex2D(IX_texture,colix,tl[tl_i2])*betas[rel_index] + tex2D(IX_texture,colix,tl[tl_i3])*gammas[rel_index];
    RIY[idx] = tex2D(IY_texture,colix,tl[tl_i1])*alphas[rel_index] + tex2D(IY_texture,colix,tl[tl_i2])*betas[rel_index] + tex2D(IY_texture,colix,tl[tl_i3])*gammas[rel_index];
    RIZ[idx] = tex2D(IZ_texture,colix,tl[tl_i1])*alphas[rel_index] + tex2D(IZ_texture,colix,tl[tl_i2])*betas[rel_index] + tex2D(IZ_texture,colix,tl[tl_i3])*gammas[rel_index];
}



__global__ void render_texture_basis_texture(
        const float* __restrict__ alphas, const float* __restrict__ betas, const float* __restrict__ gammas,
        const uint* __restrict__ indices, const int N1, const ushort* __restrict__ tl,
        float* __restrict__ RTEX, const ushort* __restrict__ triangle_idx,
        const ushort Kbeta)
{
    const int rowix = blockIdx.x;
    const int colix = threadIdx.x;

    const int rel_index = indices[rowix];

    //! Important! We fill REX, ... in a ROW-MAJOR order. This way it will be easier to extract a submatrix of REX that ignores the bottom of REX
    const int idx = threadIdx.x + Kbeta*blockIdx.x;

    const int tl_i1 = triangle_idx[rel_index];
    const int tl_i2 = tl_i1 + N_TRIANGLES;
    const int tl_i3 = tl_i2 + N_TRIANGLES;

    RTEX[idx] = tex2D(TEX_texture,colix,tl[tl_i1])*alphas[rel_index] + tex2D(TEX_texture,colix,tl[tl_i2])*betas[rel_index] + tex2D(TEX_texture,colix,tl[tl_i3])*gammas[rel_index];
}








std::vector<std::string> glob(const std::string& pattern) {
    using namespace std;

    // glob struct resides on the stack
    glob_t glob_result;
    memset(&glob_result, 0, sizeof(glob_result));

    // do the glob operation
    int return_value = glob(pattern.c_str(), GLOB_TILDE, NULL, &glob_result);
    if(return_value != 0) {
        globfree(&glob_result);
        stringstream ss;
        ss << "glob() failed with return_value " << return_value << endl;
        throw std::runtime_error(ss.str());
    }

    // collect all the filenames into a std::list<std::string>
    vector<string> filenames;
    for(size_t i = 0; i < glob_result.gl_pathc; ++i) {
        filenames.push_back(string(glob_result.gl_pathv[i]));
    }

    // cleanup
    globfree(&glob_result);

    // done
    return filenames;
}


















/*
 *
 *
import numpy


def gaussian_kernel(width = 7, sigma = 0.5):
    assert width == numpy.floor(width),  'argument width should be an integer!'
    radius = (width - 1)/2.0
    x = numpy.linspace(-radius,  radius,  width)
    x = numpy.float32(x)
    sigma = numpy.float32(sigma)
    filterx = x*x / (2 * sigma * sigma)
    filterx = numpy.exp(-1 * filterx)
    assert filterx.sum()>0,  'something very wrong if gaussian kernel sums to zero!'
    filterx /= filterx.sum()
    return filterx

f = gaussian_kernel()
 *
 */





