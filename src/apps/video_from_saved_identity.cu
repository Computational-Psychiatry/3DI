#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "config.h"
#include <experimental/filesystem>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/videoio.hpp>

#include <string>
#include <cmath>
#include <deque>

#include <vector>
#include <stdio.h>
#include <numeric>
#include <random>
#include <algorithm>    // std::shuffle
#include <set>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "constants.h"
#include "renderer.h"

#include "derivative_computer.h"
#include "logbarrier_initializer.h"

#include "model_fitter.h"
#include "funcs.h"
#include "newfuncs.h"
#include "rotation_header.h"
#include "Optimizer.h"
#include "camera.h"
#include "solver.h"
#include "preprocessing.h"
#include "video_fitter.h"

#include <glob.h> // glob(), globfree()
#include <string.h> // memset()
#include <stdexcept>
#include <sstream>

#include <opencv2/dnn.hpp>

#ifdef VISUALIZE_3D
#include "GLfuncs.h"

#endif


using std::vector;


int main(int argc, char** argv)
{
    if (argc < 2) {
        std::cout << "You need at least one argument -- the filepath for the input video" << std::endl;
        return -1;
    }

    std::string video_path(argv[1]);
    std::string landmarks_path(argv[2]);
    std::string config_filepath(argv[3]);
    config::set_params_from_YAML_file(config_filepath);
    if (!config::check_all_necessary_files())
        return 1;


    if (argc < 3) {
        std::cout << "we need at least 2 arguments (the 2nd needs to be output dir)" << std::endl;
    }

    Camera cam0;
    float field_of_view = 40;

    //    std::string calibration_path = "./models/cameras/TreeCam_1041a.txt";
    std::string calibration_path("");
    if (argc >= 5) {
        if (!is_float(argv[4]))
        {
            calibration_path = argv[4];
            cam0.init(calibration_path);
        } else {
            field_of_view = std::stof(argv[4]);
        }
    }


    double FPSvid=30;
    {
        cv::VideoCapture tmpCap(video_path);
    	FPSvid = tmpCap.get(cv::CAP_PROP_FPS);
    }


    std::string shpPath(argv[5]);
    std::string texPath(argv[6]);

    std::string exp_path(argv[7]);
    std::string pose_path(argv[8]);
    std::string illum_path(argv[9]);

    if (!cam0.initialized)
    {
        cv::VideoCapture tmpCap(video_path);

        int video_width = tmpCap.get(cv::CAP_PROP_FRAME_WIDTH);
        int video_height = tmpCap.get(cv::CAP_PROP_FRAME_HEIGHT);

//        std::cout << video_width << '\t' << video_height << std::endl;

        tmpCap.release();

        float cam_cx = video_width/2.0;
        float cam_cy = video_height/2.0;

//        double angle_x = 120.0f*M_PI/180.0; // angle in radians
        double angle_x = field_of_view*M_PI/180.0; // angle in radians
        double angle_y = angle_x; //60.0f*M_PI/180.0; //(cam_cy/cam_cx)*angle_x;

        float cam_alphax = cam_cx/(tan(angle_x/2.0));
        float cam_alphay = cam_alphax; //cam_cy/(tan(angle_y/2.0));

        cam0.init(cam_alphax, cam_alphay, cam_cx, cam_cy, false);
    }

    std::vector<std::vector<float> > selected_frame_xps, selected_frame_yps;
    std::vector<std::vector<float> > selected_frame_xranges, selected_frame_yranges;
    std::vector<cv::Mat> selected_frames;

    float *h_X0, *h_Y0, *h_Z0, *h_tex_mu;

    h_X0 = (float*)malloc( config::NPTS*sizeof(float) );
    h_Y0 = (float*)malloc( config::NPTS*sizeof(float) );
    h_Z0 = (float*)malloc( config::NPTS*sizeof(float) );
    h_tex_mu = (float*)malloc( config::NPTS*sizeof(float) );
    int min_x(0), max_x(0), min_y(0), max_y(0);

    std::cout << "read identity" << std::endl;

    std::vector< std::vector<float> > id = read2DVectorFromFile<float>(shpPath,  config::NPTS, 3);
    std::vector< std::vector<float> > tex = read2DVectorFromFile<float>(texPath ,  config::NPTS, 1);

    for (size_t pi=0; pi<config::NPTS; ++pi)
    {
        h_X0[pi] = id[pi][0];
        h_Y0[pi] = id[pi][1];
        h_Z0[pi] = id[pi][2];
        h_tex_mu[pi] = tex[pi][0];
    }

    cam0.update_camera(1.0f);

    VideoFitter vf(cam0, 0, 0, config::K_EPSILON,
                   0, 0, config::K_EPSILON_L, config::TIME_T,
                    config::USE_TEMP_SMOOTHING, config::USE_EXP_REGULARIZATION,
                    h_X0, h_Y0, h_Z0, h_tex_mu);

    // Bind texture memories
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipChannelFormatDesc desc2 = hipCreateChannelDesc<float>();
    hipChannelFormatDesc desc3 = hipCreateChannelDesc<float>();

    LandmarkData ld(landmarks_path);

    VideoOutput vid_out = vf.fit_video_frames_auto(video_path, ld, &min_x, &max_x, &min_y, &max_y);
    vid_out.save_expressions(exp_path);
    vid_out.save_poses(pose_path, &vf.ov, &vf.rc);
    vid_out.save_illums(illum_path);

    free(h_X0);
    free(h_Y0);
    free(h_Z0);
    free(h_tex_mu);
}









std::vector<std::string> glob(const std::string& pattern) {
    using namespace std;

    // glob struct resides on the stack
    glob_t glob_result;
    memset(&glob_result, 0, sizeof(glob_result));

    // do the glob operation
    int return_value = glob(pattern.c_str(), GLOB_TILDE, NULL, &glob_result);
    if(return_value != 0) {
        globfree(&glob_result);
        stringstream ss;
        ss << "glob() failed with return_value " << return_value << endl;
        throw std::runtime_error(ss.str());
    }

    // collect all the filenames into a std::list<std::string>
    vector<string> filenames;
    for(size_t i = 0; i < glob_result.gl_pathc; ++i) {
        filenames.push_back(string(glob_result.gl_pathv[i]));
    }

    // cleanup
    globfree(&glob_result);

    // done
    return filenames;
}


















/*
 *
 *
import numpy


def gaussian_kernel(width = 7, sigma = 0.5):
    assert width == numpy.floor(width),  'argument width should be an integer!'
    radius = (width - 1)/2.0
    x = numpy.linspace(-radius,  radius,  width)
    x = numpy.float32(x)
    sigma = numpy.float32(sigma)
    filterx = x*x / (2 * sigma * sigma)
    filterx = numpy.exp(-1 * filterx)
    assert filterx.sum()>0,  'something very wrong if gaussian kernel sums to zero!'
    filterx /= filterx.sum()
    return filterx

f = gaussian_kernel()
 *
 */





