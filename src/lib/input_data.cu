#include "hip/hip_runtime.h"
#include "input_data.h"
#include "constants.h"
#include "preprocessing.h"
#include "funcs.h"

void InputData::add_data(const cv::Mat& frame, const std::vector<float>& xp, const std::vector<float> &yp, size_t fi, float face_size)
{
    if (frames.size() == T)
    {
        /*
        frames.clear();
        xp_origs.clear();
        yp_origs.clear();
        abs_frame_ids.clear();
        face_sizes.clear();
        */
        frames.pop_front();
        xp_origs.pop_front();
        yp_origs.pop_front();
        abs_frame_ids.pop_front();
        face_sizes.pop_front();
    }

    frames.push_back(frame);
    xp_origs.push_back(xp);
    yp_origs.push_back(yp);
    abs_frame_ids.push_back(fi);
    face_sizes.push_back(face_size);
}


void InputData::get_resized_landmarks(size_t rel_frame_id, const float resize_coef, float* xp, float *yp)
{
    // @@@ the landmarks will probably be already resized in the data structure
    for (int i=0; i<NLANDMARKS_51; ++i)
    {
        xp[i] = resize_coef*(xp_origs[rel_frame_id][i]);
        yp[i] = resize_coef*(yp_origs[rel_frame_id][i]);
    }
}


void InputData::get_resized_frame(size_t rel_frame_id, const float resize_coef, cv::Mat& frame_dst)
{
    cv::cvtColor(frames[rel_frame_id], frame_dst, cv::COLOR_BGR2GRAY);

    frame_dst.convertTo(frame_dst, CV_32FC1);
    frame_dst = frame_dst/255.0f;

    // @@@ probably needs to be done for all cams
    cv::resize(frame_dst, frame_dst, cv::Size(), resize_coef, resize_coef);
    cv::copyMakeBorder(frame_dst, frame_dst, 0, DIMY, 0, DIMX, cv::BORDER_CONSTANT, 0);
}


void InputData::clear()
{
    frames.clear();
    xp_origs.clear();
    yp_origs.clear();
    abs_frame_ids.clear();
}


LandmarkData::LandmarkData()
{
    LandmarkData::check_CUDA(LANDMARK_DETECTOR_BACKEND, LANDMARK_DETECTOR_TARGET);
}

LandmarkData::LandmarkData(const std::string& landmarks_path)
{
    LandmarkData::check_CUDA(LANDMARK_DETECTOR_BACKEND, LANDMARK_DETECTOR_TARGET);

    init_from_txtfile(landmarks_path);
}


LandmarkData::LandmarkData(const std::string &video_path, const std::string &faces_path, const std::string& landmarks_path)
{
    LandmarkData::check_CUDA(LANDMARK_DETECTOR_BACKEND, LANDMARK_DETECTOR_TARGET);

    vector<vector<float> > face_rects;
    if (std::experimental::filesystem::exists(faces_path))
        face_rects = read2DVectorFromFile_unknown_size<float>(faces_path);
    else
        face_rects = detect_faces(video_path, faces_path);

    vector<vector<float> > all_lmks;
    if (std::experimental::filesystem::exists(landmarks_path))
        all_lmks = read2DVectorFromFile_unknown_size<float>(landmarks_path);
    else
        all_lmks = detect_landmarks(video_path, face_rects, landmarks_path);

    fill_xpypvec(all_lmks);
}


void LandmarkData::init_from_txtfile(const std::string &landmarks_path)
{
    vector<vector<float> > all_lmks = read2DVectorFromFile_unknown_size<float>(landmarks_path);
    fill_xpypvec(all_lmks);

}

void LandmarkData::fill_xpypvec(vector<vector<float> > &all_lmks)
{
    size_t T = all_lmks.size();
    for (size_t t=0; t<T; ++t)
    {
        vector<float> xp_vec, yp_vec;

        for (size_t i=0; i<NLANDMARKS_51; ++i)
        {
            xp_vec.push_back(all_lmks[t][2*i]);
            yp_vec.push_back(all_lmks[t][2*i+1]);
        }

        xp_vecs.push_back(xp_vec);
        yp_vecs.push_back(yp_vec);
    }
}

int LandmarkData::get_face_size(size_t t)
{
    vector<float> xp_vec = get_xpvec(t);
    vector<float> yp_vec = get_ypvec(t);

    //    cv::waitKey(0);
    int cur_xmin = (int) *std::min_element(xp_vec.begin(), xp_vec.end());
    int cur_xmax = (int) *std::max_element(xp_vec.begin(), xp_vec.end());

    int cur_ymin = (int) *std::min_element(yp_vec.begin(), yp_vec.end());
    int cur_ymax = (int) *std::max_element(yp_vec.begin(), yp_vec.end());

    int face_width = cur_xmax-cur_xmin;
    int face_height = cur_ymax-cur_ymin;
    return (float) std::max<int>(face_width, face_height);
}


bool LandmarkData::check_CUDA(cv::dnn::Backend& LANDMARK_DETECTOR_BACKEND, cv::dnn::Target& LANDMARK_DETECTOR_TARGET)
{
    const std::string caffeConfigFile = config::FACE_DETECTOR_DPATH;
    const std::string caffeWeightFile = config::FACE_DETECTOR_MPATH;
    cv::dnn::Net landmark_net = cv::dnn::readNetFromTensorflow(config::LANDMARK_MPATH);

    landmark_net.setPreferableBackend(cv::dnn::DNN_BACKEND_CUDA);
    landmark_net.setPreferableTarget(cv::dnn::DNN_TARGET_CUDA);

    cv::Mat im_cropped(256, 256, CV_32FC3, cv::Scalar::all(1) );
    cv::Mat net_input = cv::dnn::blobFromImage(im_cropped);
    landmark_net.setInput(net_input);

    //cv::Mat netOut = landmark_net.forward().clone();
    try {
        cv::Mat netOut = landmark_net.forward().clone();
    } catch (cv::Exception) {
        std::cout << "OpenCV does not seem to have been install with CUDA support. If you did install with CUDA support, you may have compiled it with the wrong CUDA_ARCH_BIN parameter. " << std::endl <<
                     "Consider re-compiling OpenCV, and during compilation make sure that the CUDA_ARCH_BIN is the correct one for your device. (Our github repository contains a wiki page with compilation instructions.  )" << std::endl <<
                     "Without CUDA support, landmark detection will be very slow." << std::endl;
        LANDMARK_DETECTOR_BACKEND = cv::dnn::DNN_BACKEND_OPENCV;
        LANDMARK_DETECTOR_TARGET = cv::dnn::DNN_TARGET_CPU;
        return false;
    }
    return true;
}

vector<vector<float> > LandmarkData::detect_faces(const std::string& filepath, const std::string& rects_filepath)
{
    const std::string caffeConfigFile = config::FACE_DETECTOR_DPATH;
    const std::string caffeWeightFile = config::FACE_DETECTOR_MPATH;

    std::string framework = "caffe";

    cv::dnn::Net detection_net = cv::dnn::readNetFromCaffe(caffeConfigFile, caffeWeightFile);
    detection_net.setPreferableBackend(LANDMARK_DETECTOR_BACKEND);
    detection_net.setPreferableTarget(LANDMARK_DETECTOR_TARGET);

    cv::VideoWriter video_out;

    cv::VideoCapture capture(filepath);

    if( !capture.isOpened() )
        throw "Error when reading steam_avi";

    cv::Mat frame;

    vector<float> face_sizes;
    int idx = 0;
    cv::Rect ROI(-1, -1, -1, -1);

    vector<vector<float> > face_rects;
    while (true) {
        idx++;

        std::cout << "Processing frame #" << idx << '\r' << std::flush;
        vector<float> xp_vec, yp_vec;
        vector<float> xrange, yrange;

        capture >> frame;

        if (frame.empty())
            break;

        if (idx < 0)
            continue;

        double face_confidence;
        cv::Rect face_rect = detect_face_opencv(detection_net, framework, frame, &ROI, &face_confidence, true);
        face_rects.push_back(vector<float>({(float)face_rect.x, (float)face_rect.y, (float)face_rect.width, (float)face_rect.height}));

        if (idx >= config::MAX_VID_FRAMES_TO_PROCESS)
            break;
    }

    write_2d_vector<float>(rects_filepath, face_rects);

    return face_rects;
}


vector<vector<float> > LandmarkData::detect_landmarks(const std::string &video_filepath, const vector<vector<float> > &face_rects, const std::string &landmarks_filepath)
{
    vector<vector<float> > all_lmks;

    const std::string caffeConfigFile = config::FACE_DETECTOR_DPATH;
    const std::string caffeWeightFile = config::FACE_DETECTOR_MPATH;

    std::string device = "CPU";
    std::string framework = "caffe";

    cv::dnn::Net detection_net = cv::dnn::readNetFromCaffe(caffeConfigFile, caffeWeightFile);

    cv::dnn::Net landmark_net = cv::dnn::readNetFromTensorflow(config::LANDMARK_MPATH);
    landmark_net.setPreferableBackend(LANDMARK_DETECTOR_BACKEND);
    landmark_net.setPreferableTarget(LANDMARK_DETECTOR_TARGET);

    cv::dnn::Net leye_net = cv::dnn::readNetFromTensorflow(config::LANDMARK_LEYE_MPATH);
    leye_net.setPreferableBackend(LANDMARK_DETECTOR_BACKEND);
    leye_net.setPreferableTarget(LANDMARK_DETECTOR_TARGET);

    cv::dnn::Net reye_net = cv::dnn::readNetFromTensorflow(config::LANDMARK_REYE_MPATH);
    reye_net.setPreferableBackend(LANDMARK_DETECTOR_BACKEND);
    reye_net.setPreferableTarget(LANDMARK_DETECTOR_TARGET);

    cv::dnn::Net mouth_net = cv::dnn::readNetFromTensorflow(config::LANDMARK_MOUTH_MPATH);
    mouth_net.setPreferableBackend(LANDMARK_DETECTOR_BACKEND);
    mouth_net.setPreferableTarget(LANDMARK_DETECTOR_TARGET);

    cv::dnn::Net correction_net = cv::dnn::readNetFromTensorflow(config::LANDMARK_CORRECTION_MPATH);

    cv::VideoCapture capture(video_filepath);

    if( !capture.isOpened() )
        throw "Error when reading steam_avi";

    cv::Mat frame;


    int Nframes = std::min<int>(config::MAX_VID_FRAMES_TO_PROCESS, (int) capture.get(cv::CAP_PROP_FRAME_COUNT));
    int idx = 0;
    cv::Rect ROI(-1, -1, -1, -1);
    while (true) {
        idx++;

        std::cout << "Processing frame #" << idx << "/" << Nframes << '\r' << std::flush;

        vector<float> xp_vec, yp_vec;
        vector<float> xrange, yrange;

        if (config::PRINT_DEBUG) {
            if (idx % config::PRINT_EVERY_N_FRAMES == 0)
                std::cout << "Processing frame# " << idx << std::endl;
        }

        capture >> frame;

        if (frame.empty())
            break;

        if (idx >= config::MAX_VID_FRAMES_TO_PROCESS)
            break;

        float face_size;

        double face_confidence(0.99);

        if (idx-1 >= face_rects.size()) {
            std::cout << "WARNING: Looks like there are not enough face rectangles during landmark detection; breaking" << std::endl;
        }

        cv::Rect face_rect(face_rects[idx-1][0], face_rects[idx-1][1], face_rects[idx-1][2], face_rects[idx-1][3]);

        try {
            if (face_rect.width > 10) {
                detect_landmarks_opencv(face_rect, face_confidence, landmark_net, leye_net, reye_net, mouth_net, correction_net, frame,
                                        xp_vec, yp_vec, face_size, xrange, yrange, config::USE_LOCAL_MODELS, false);
            }
        } catch (std::exception& e)
        {
            std::cout << "Problem with landmark detection at frame " << idx << std::endl;
        }

        vector<float> lmks_combined;
        for (size_t i=0; i<NLANDMARKS_51; ++i)
        {
            if (xp_vec.size() == 51 && yp_vec.size() == 51)
            {
                lmks_combined.push_back(xp_vec[i]);
                lmks_combined.push_back(yp_vec[i]);
            }
            else
            {
                lmks_combined.push_back(0);
                lmks_combined.push_back(0);
            }
        }

        all_lmks.push_back(lmks_combined);

        if (face_size == -1.0f)
            continue;

        if (idx > config::MAX_VID_FRAMES_TO_PROCESS)
            break;
    }

    std::cout << std::endl;

    write_2d_vector<float>(landmarks_filepath, all_lmks);

    return all_lmks;
}




